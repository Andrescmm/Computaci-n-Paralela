#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#if defined(NDEBUG)
#define CUDA_CHECK(X) (X)
#else
#define CUDA_CHECK(X) do{\
	(X);\
	hipError_t e = hipGetLastError();\
	if(hipSuccess != e){\
		printf("cuda failure %s at %s : %d",hipGetErrorString(e), __FILE__, __LINE__);\
		exit(1);\
	}\
}while(0)
#endif



__global__ void mulKernel(int* c, const int* a, const int* b, const int WIDTH) {
	int x = threadIdx.x;
	int y = threadIdx.y;
	int i = y * WIDTH + x;

	int sum = 0;
	for (int k = 0; k < WIDTH; k++) {
		sum += a[y * WIDTH + k] * b[k * WIDTH + x];
	}
	c[i] = sum;
}

int main()
{
    int WIDTH;
    
    printf("\n INGRESAR TAMAÑO DE LAS MATRICES:");
    scanf("%d",&WIDTH);
	
	int a[WIDTH][WIDTH];
	int b[WIDTH][WIDTH];
	int c[WIDTH][WIDTH] = { 0 };

	for (int y = 0; y < WIDTH; y++) {
		for (int x = 0; x < WIDTH; x++) {
			a[y][x] = rand() % 20;
			b[y][x] = rand() % 20;
		}
	}
/*
    //imprimiendo matriz A
    printf("VALORES DE MATRIZ A \n");
	for (int y = 0; y < WIDTH; y++)
	{
		for (int x = 0; x < WIDTH; x++)
		{
			printf("%5d", a[y][x]);
		}
		printf("\n");
	}
	
	printf("\n\n VALORES DE MATRIZ B \n");
	for (int y = 0; y < WIDTH; y++)
	{
		for (int x = 0; x < WIDTH; x++)
		{
			printf("%5d", b[y][x]);
		}
		printf("\n\n");
	}
*/	
	//device side data
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;

	//allocate device memory
	hipMalloc((void**)&dev_a, WIDTH * WIDTH * sizeof(int));
	hipMalloc((void**)&dev_b, WIDTH * WIDTH * sizeof(int));
	hipMalloc((void**)&dev_c, WIDTH * WIDTH * sizeof(int));
    
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    
	//copy from host to device
	hipMemcpy(dev_a, a, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);	//dev_a = a
	hipMemcpy(dev_b, b, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);	//dev_b = b

	//launch a kernel on the GPU with one thread for each element
	dim3 dimBlock(WIDTH, WIDTH, 1);	//x,y,z
	hipEventRecord(start);
	mulKernel << <1, dimBlock >> > (dev_c, dev_a, dev_b, WIDTH);
	hipEventRecord(stop);
	CUDA_CHECK(hipPeekAtLastError());

	//copy from device to host
	hipMemcpy(c, dev_c, WIDTH * WIDTH * sizeof(int), hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

	//free device memory
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

/*	//print the result
	printf("RESULTADO DE MULTIPLICACION -> MATRIZ C \n");
	for (int y = 0; y < WIDTH; y++) {
		for (int x = 0; x < WIDTH; x++) {
			printf("%10d", c[y][x]);
		}
		printf("\n");
	}
*/	
	printf("%fn <-TIME ", milliseconds);
	return 0;
}

